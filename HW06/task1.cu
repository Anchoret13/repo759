#include <iostream>
#include <random>
#include <chrono>
#include "matmul.cuh"

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " n threads_per_block" << std::endl;
        return 1;
    }

    size_t n = std::stoi(argv[1]);
    unsigned int threads_per_block = std::stoi(argv[2]);

    if (n <= 0 || threads_per_block <= 0) {
        std::cerr << "Error: Matrix size and threads_per_block must be positive integers" << std::endl;
        return 1;
    }

    float* h_A = new float[n * n];
    float* h_B = new float[n * n];
    float* h_C = new float[n * n];

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

    for (size_t i = 0; i < n * n; i++) {
        h_A[i] = dist(gen);
        h_B[i] = dist(gen);
    }

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));

    hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    matmul(d_A, d_B, d_C, n, threads_per_block);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << h_C[n * n - 1] << std::endl;
    std::cout << milliseconds << std::endl;

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}