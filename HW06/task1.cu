#include <iostream>
#include <random>
#include "matmul.cuh"

int main(int argc, char* argv[]) {
    try {
        if (argc != 3) {
            std::cerr << "Usage: " << argv[0] << " n threads_per_block" << std::endl;
            return 1;
        }

        size_t n = std::stoi(argv[1]);
        unsigned int threads_per_block = std::stoi(argv[2]);

        std::cout << "Starting with n=" << n << ", threads_per_block=" << threads_per_block << std::endl;

        float* h_A = new float[n * n];
        float* h_B = new float[n * n];
        float* h_C = new float[n * n];

        std::random_device rd;
        std::mt19937 gen(rd());
        std::uniform_real_distribution<float> dist(-1.0f, 1.0f);

        for (size_t i = 0; i < n * n; i++) {
            h_A[i] = dist(gen);
            h_B[i] = dist(gen);
        }
        
        // Print a few values to verify initialization
        std::cout << "First few values of A: " << h_A[0] << ", " << h_A[1] << std::endl;
        std::cout << "First few values of B: " << h_B[0] << ", " << h_B[1] << std::endl;

        float *d_A, *d_B, *d_C;
        hipError_t err;
        
        err = hipMalloc(&d_A, n * n * sizeof(float));
        if (err != hipSuccess) {
            std::cerr << "Failed to allocate d_A: " << hipGetErrorString(err) << std::endl;
            return 1;
        }
        
        err = hipMalloc(&d_B, n * n * sizeof(float));
        if (err != hipSuccess) {
            std::cerr << "Failed to allocate d_B: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            return 1;
        }
        
        err = hipMalloc(&d_C, n * n * sizeof(float));
        if (err != hipSuccess) {
            std::cerr << "Failed to allocate d_C: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            hipFree(d_B);
            return 1;
        }

        err = hipMemcpy(d_A, h_A, n * n * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Failed to copy A to device: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            return 1;
        }
        
        err = hipMemcpy(d_B, h_B, n * n * sizeof(float), hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            std::cerr << "Failed to copy B to device: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            return 1;
        }

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        matmul(d_A, d_B, d_C, n, threads_per_block);
        
        err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "Kernel launch failed: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            return 1;
        }

        hipEventRecord(stop);
        hipEventSynchronize(stop);

        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        err = hipMemcpy(h_C, d_C, n * n * sizeof(float), hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            std::cerr << "Failed to copy C from device: " << hipGetErrorString(err) << std::endl;
            hipFree(d_A);
            hipFree(d_B);
            hipFree(d_C);
            return 1;
        }

        std::cout << "Result calculation complete. Last element: " << h_C[n * n - 1] << std::endl;
        std::cout << h_C[n * n - 1] << std::endl;
        std::cout << milliseconds << std::endl;

        delete[] h_A;
        delete[] h_B;
        delete[] h_C;
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        hipEventDestroy(start);
        hipEventDestroy(stop);
    }
    catch (const std::exception& e) {
        std::cerr << "Exception caught: " << e.what() << std::endl;
        return 1;
    }
    catch (...) {
        std::cerr << "Unknown exception caught" << std::endl;
        return 1;
    }

    return 0;
}